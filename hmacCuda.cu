#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "hmac.h"
#include "sha1.h"

#define KEY_LEN 20
#define MESSAGE_LEN 34
// TODO: maybe 20
#define DIGEST_LEN 20

int main()
{
    unsigned char *key, *message, *digest;
    unsigned char *d_key, *d_message, *d_digest;

    // Allocate host memory
    key = (unsigned char*)malloc(sizeof(unsigned char) * KEY_LEN);
    message = (unsigned char*)malloc(sizeof(unsigned char) * MESSAGE_LEN);
    digest = (unsigned char*)malloc(sizeof(unsigned char) * DIGEST_LEN);

    // Initialize host arrays
    for (int i = 0; i < KEY_LEN; i++)
		  key[i] = i;
    
    char *messageStr = "Sample message for keylen<blocklen";\
    for (int i = 0; i < MESSAGE_LEN; i++)
      message[i] = messageStr[i];

    // Allocate device memory
    hipMalloc((void**)&d_key, sizeof(unsigned char) * KEY_LEN);
    hipMalloc((void**)&d_message, sizeof(unsigned char) * MESSAGE_LEN);
    hipMalloc((void**)&d_digest, sizeof(unsigned char) * DIGEST_LEN);

    // Transfer data from host to device memory
    hipMemcpy(d_key, key, sizeof(unsigned char) * KEY_LEN, hipMemcpyHostToDevice);
    hipMemcpy(d_message, message, sizeof(unsigned char) * MESSAGE_LEN, hipMemcpyHostToDevice);

    // Executing kernel
    __hmac_sha1<<<1,1>>>(d_key, KEY_LEN, d_message, MESSAGE_LEN, d_digest);

    // Transfer data back to host memory
    hipMemcpy(digest, d_digest, sizeof(unsigned char) * DIGEST_LEN, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < DIGEST_LEN; i++){
        printf("%02x", digest[i]);
    }
    printf("\n");

    // Deallocate device memory
    hipFree(d_key);
    hipFree(d_message);
    hipFree(d_digest);

    // Deallocate host memory
    free(key); 
    free(message); 
    free(digest);
    return 0;
}
